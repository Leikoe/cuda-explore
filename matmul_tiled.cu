
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <time.h>
#include <stdint.h>

uint64_t nanos()
{
    struct timespec start;
    clock_gettime(CLOCK_MONOTONIC_RAW, &start);
    return (uint64_t)start.tv_sec * 1000000000 + (uint64_t)start.tv_nsec;
}

#define N 1024
#define CEIL_DIV(a,b) ((a+b-1)/b)
#define WARP_SIZE 32
#define BLOCK_SIZE 16

__global__ void matmul(float *a, float *b, float *c, int n)
{
	int row = blockIdx.y;
	int col = blockIdx.x;
    int thread_row = threadIdx.x / n;
    int thread_col = threadIdx.x % n;

    if (row * blockDim.y + thread_row >= n or col * blockDim.x + thread_col)
    {
        return;
    }

	__shared__ float tile_a[BLOCK_SIZE * BLOCK_SIZE];
	__shared__ float tile_b[BLOCK_SIZE * BLOCK_SIZE];

    float acc = 0.0f;
    for (int block_idx = 0; block_idx < n; block_idx+=BLOCK_SIZE)
    {
		tile_a[thread_row * BLOCK_SIZE + thread_col] = a[row * n + col];
		tile_b[thread_row * BLOCK_SIZE + thread_col] = b[row * n + col];
		
		__syncthreads();  // wait for all the threads in the warp to load their item of the block into the block (smem)

		a += block_idx * BLOCK_SIZE * n;
		b += block_idx * BLOCK_SIZE * n;

		for (int k = 0; k < BLOCK_SIZE; k++) {
			acc += tile_a[thread_row * BLOCK_SIZE + k] * tile_b[k * BLOCK_SIZE + thread_col];
		}

		__syncthreads();  // we don't want to change the tiles in smem while some threads are still accumulating
		
    }

    c[row * n + col] = acc;
}

int main()
{
    float *a = (float *)malloc(N * N * sizeof(float));
    float *b = (float *)malloc(N * N * sizeof(float));
    float *c = (float *)malloc(N * N * sizeof(float));

    float *d_a, *d_b, *d_c;
    hipMalloc(&d_a, N * N * sizeof(float));
    hipMalloc(&d_b, N * N * sizeof(float));
    hipMalloc(&d_c, N * N * sizeof(float));

    // fill a & b and zero out c
    for (int i = 0; i < (N * N); i++)
    {
        a[i] = 1.0f;
        b[i] = 2.0f;
        c[i] = 0.0f;
    }

    hipMemcpy(d_a, a, N * N * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_b, b, N * N * sizeof(float), hipMemcpyHostToDevice);

	dim3 grid_dim(CEIL_DIV(N, WARP_SIZE), CEIL_DIV(N, WARP_SIZE));
	dim3 block_dim(WARP_SIZE * WARP_SIZE);
    printf("LAUNCHING with grid_dim: (%d, %d) and block_dim: (%d, %d)\n", grid_dim.x, grid_dim.y, block_dim.x, block_dim.y);

    uint64_t start = nanos();
    matmul<<<grid_dim, block_dim>>>(d_a, d_b, d_c, N);
    hipDeviceSynchronize();
    uint64_t end = nanos();

    hipMemcpy(c, d_c, N * N * sizeof(float), hipMemcpyDeviceToHost);

    double gflop = (2.0 * N * N * N) * 1e-9;
    double s = (end - start) * 1e-9;
    printf("%f GFLOP/S -- %.2f ms\n", gflop / s, s * 1e3);

    {
		// compute naive reference matmul on cpu
    	printf("Computing reference matmul result on cpu\n");
		float *reference_c = (float *)malloc(N * N * sizeof(float));
    	for (int i = 0; i < N; i++)
    	{

    		for (int j = 0; j < N; j++)
    		{
				float acc = 0.0f;
				for (int k = 0; k < N; k++)
				{
					acc += a[i * N + k] * b[k * N + j];
				}
				reference_c[i * N + j] = acc;
			}
    	}

		// check each item
		printf("Comparing reference result with gpu result\n");
		for (int i = 0; i < N; i++)
		{
			for (int j = 0; j < N; j++)
			{
				if (c[i * N + j] != reference_c[i * N + j])
				{
					printf("ERROR at i=%d j=%d (should be %f, is %f)\n", i, j, reference_c[i * N + j], c[i * N + j]);
					exit(1);
				}
			}
		}
		free(reference_c);
		printf("ALL GOOD\n");
    }
    
    hipFree(d_a);
    hipFree(d_b);
    hipFree(d_c);
    free(a);
    free(b);
    free(c);
}

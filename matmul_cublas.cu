#include <cstdio>
#include <hipblas.h>
#include <hip/hip_runtime.h>
#include <stdio.h>
#include "utils.cu"

#define N 4096

int main()
{
    hipblasStatus_t stat;   // cuBLAS functions status
    hipblasHandle_t handle; // cuBLAS context

    srand(time(NULL));

    CudaDeviceInfo();

    float *a = (float *)malloc(N * N * sizeof(float));
    float *b = (float *)malloc(N * N * sizeof(float));
    float *c = (float *)malloc(N * N * sizeof(float));

    float *d_a, *d_b, *d_c;
    hipMalloc(&d_a, N * N * sizeof(float));
    hipMalloc(&d_b, N * N * sizeof(float));
    hipMalloc(&d_c, N * N * sizeof(float));

    stat = hipblasCreate(&handle); // initialize CUBLAS context

    // fill a & b and zero out c
    matrix_random(a, N*N);
    matrix_random(b, N*N);
    matrix_zeros(c, N*N);

    hipMemcpy(d_a, a, N * N * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_b, b, N * N * sizeof(float), hipMemcpyHostToDevice);
	
    float alpha = 1.0;
    float beta = 1.0;

    uint64_t start = nanos();
    stat = hipblasSgemm(handle, HIPBLAS_OP_N, HIPBLAS_OP_N, N, N, N, &alpha, d_b, N,
                     d_a, N, &beta, d_c, N);

    hipDeviceSynchronize();
    uint64_t end = nanos();

    hipMemcpy(c, d_c, N * N * sizeof(float), hipMemcpyDeviceToHost);

    double gflop = (2.0 * N * N * N) * 1e-9;
    double s = (end - start) * 1e-9;
    printf("%f GFLOP/S -- %.2f ms\n", gflop / s, s * 1e3);

    // {
	// 	// compute naive reference matmul on cpu
    // 	printf("Computing reference matmul result on cpu\n");
	// 	float *reference_c = (float *)malloc(N * N * sizeof(float));
    // 	matmul_c(a, b, reference_c, N);

	// 	// check each item
	// 	matrix_eq(reference_c, c, N);
	// 	free(reference_c);
	// 	printf("ALL GOOD\n");
    // }
    
    hipFree(d_a);
    hipFree(d_b);
    hipFree(d_c);
    hipblasDestroy(handle); // destroy CUBLAS context
    free(a);
    free(b);
    free(c);
}

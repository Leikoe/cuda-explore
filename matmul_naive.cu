#include "hip/hip_runtime.h"
#include <stdio.h>
#include <time.h>
#include <stdint.h>
#include <assert.h>
#include <hip/hip_fp16.h>
#include "utils.cu"

#define N 1024

__global__ void matmul(__half *a, __half *b, float *c, int n)
{
    int thread_id_x = blockIdx.x * blockDim.x + threadIdx.x;
    int thread_id_y = blockIdx.y * blockDim.y + threadIdx.y;

    if (thread_id_x >= N || thread_id_y >= N)
    {
        return;
    }

    float acc = 0.0f;
    for (int k = 0; k < n; k++)
    {
        acc = __half2float(__hmul(a[thread_id_y * n + k], b[k * n + thread_id_x])) + acc;
    }

    c[thread_id_y * n + thread_id_x] = acc;
}

int main()
{
    srand(time(NULL));

    float *a = (float *)malloc(N * N * sizeof(float));
    float *b = (float *)malloc(N * N * sizeof(float));
    float *c = (float *)malloc(N * N * sizeof(float));

    // fill a & b
    matrix_random_fp16valued(a, N * N);
    matrix_random_fp16valued(b, N * N);

    __half *a_h = (__half *)malloc(N * N * sizeof(__half));
    __half *b_h = (__half *)malloc(N * N * sizeof(__half));

    for (int i = 0; i < N * N; i++)
    {
        a_h[i] = __float2half(a[i]);
        b_h[i] = __float2half(b[i]);
    }

    __half *d_a, *d_b;
    float *d_c;
    hipMalloc(&d_a, N * N * sizeof(__half));
    hipMalloc(&d_b, N * N * sizeof(__half));
    hipMalloc(&d_c, N * N * sizeof(float));
    hipMemcpy(d_a, a_h, N * N * sizeof(__half), hipMemcpyHostToDevice);
    hipMemcpy(d_b, b_h, N * N * sizeof(__half), hipMemcpyHostToDevice);

    dim3 block_dim(16, 16);
    dim3 grid_dim(CEIL_DIV(N, block_dim.x), CEIL_DIV(N, block_dim.y));
    printf("LAUNCHING with grid_dim: (%d, %d) and block_dim: (%d, %d)\n", grid_dim.x, grid_dim.y, block_dim.x, block_dim.y);

    uint64_t start = nanos();
    matmul<<<grid_dim, block_dim>>>(d_a, d_b, d_c, N);
    hipDeviceSynchronize();
    uint64_t end = nanos();

    hipMemcpy(c, d_c, N * N * sizeof(float), hipMemcpyDeviceToHost);

    double gflop = (2.0 * N * N * N) * 1e-9;
    double s = (end - start) * 1e-9;
    printf("%f GFLOP/S -- %.2f ms\n", gflop / s, s * 1e3);

    {
        // compute naive reference matmul on cpu
        printf("Computing reference matmul result on cpu\n");
        float *reference_c = (float *)malloc(N * N * sizeof(float));
        matmul_c(a, b, reference_c, N);

        // check each item
        printf("Comparing reference result with gpu result\n");
        matrix_eq(reference_c, c, N);
        printf("ALL GOOD\n");
        free(reference_c);
    }

    hipFree(d_a);
    hipFree(d_b);
    hipFree(d_c);
    free(a_h);
    free(b_h);
    free(a);
    free(b);
    free(c);
}
